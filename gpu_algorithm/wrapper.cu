#include "hip/hip_runtime.h"
#include "gpu_wrapper.h"
#include "dft.h"
#include "gridder.h"

#include "correlation_gridding_traits.h"
#include "correlation_gridding_policies.h"
#include "templated_gridder.h"

#define NO_THREADS_PER_BLOCK_DIM 256

extern "C" {
    utils::timer * inversion_walltime;
    utils::timer * gridding_walltime;
    hipStream_t compute_stream;
    gridding_parameters gpu_params;
    
    double get_gridding_walltime(){
      return gridding_walltime->duration();
    }
    double get_inversion_walltime(){
      return inversion_walltime->duration();
    }
    void gridding_barrier(){
      cudaSafeCall(hipStreamSynchronize(compute_stream));
    }
    void initLibrary(gridding_parameters & params) {
        int num_devices, device;
        hipGetDeviceCount(&num_devices);
        if (num_devices > 0) {
            //get the argmax{devID}(multiProcessorCounts):
            int max_multiprocessors = 0, max_device = 0;
            for (device = 0; device < num_devices; device++) {
                hipDeviceProp_t properties;
                hipGetDeviceProperties(&properties, device);
                if (max_multiprocessors < properties.multiProcessorCount) {
                    max_multiprocessors = properties.multiProcessorCount;
                    max_device = device;
                }
            }
            hipSetDevice(max_device); //select device
            hipDeviceReset(); //ensure device is in a safe state before we begin processing

            //print some stats:
            hipDeviceProp_t properties;
            hipGetDeviceProperties(&properties, max_device);

            size_t mem_tot = 0;
            size_t mem_free = 0;
            hipMemGetInfo  (&mem_free, & mem_tot);
            printf("---------------------------------------Backend: GPU GRIDDING LIBRARY---------------------------------------\n");
            printf("%s, device %d on PCI Bus #%d, clocked at %f GHz\n",properties.name,properties.pciDeviceID,
                   properties.pciBusID,properties.clockRate / 1000000.0);
            printf("Compute capability %d.%d with %f GiB global memory (%f GiB free)\n",properties.major,
                   properties.minor,mem_tot/1024.0/1024.0/1024.0,mem_free/1024.0/1024.0/1024.0);
            printf("%d SMs are available\n",properties.multiProcessorCount);
            printf("-----------------------------------------------------------------------------------------------------------\n");
        } else 
            throw std::runtime_error("Cannot find suitable GPU device. Giving up");
	cudaSafeCall(hipStreamCreateWithFlags(&compute_stream,hipStreamNonBlocking));
	gridding_walltime = new utils::timer(compute_stream);
	inversion_walltime = new utils::timer(compute_stream);
	//alloc memory for all the arrays on the gpu at the beginning of execution...
	gpu_params = params;
	cudaSafeCall(hipMalloc((void**)&gpu_params.visibilities, sizeof(std::complex<visibility_base_type>) * params.chunk_max_row_count*params.channel_count*params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.uvw_coords, sizeof(imaging::uvw_coord<uvw_base_type>) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.reference_wavelengths, sizeof(reference_wavelengths_base_type) * params.channel_count * params.spw_count));
	cudaSafeCall(hipMemcpy(gpu_params.reference_wavelengths,params.reference_wavelengths,sizeof(reference_wavelengths_base_type) * params.channel_count * params.spw_count,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.enabled_channels, sizeof(bool) * params.channel_count * params.spw_count));
	cudaSafeCall(hipMemcpy(gpu_params.enabled_channels,params.enabled_channels, sizeof(bool) * params.channel_count * params.spw_count,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.channel_grid_indicies, sizeof(size_t) * params.channel_count * params.spw_count));
	cudaSafeCall(hipMemcpy(gpu_params.channel_grid_indicies,params.channel_grid_indicies, sizeof(size_t) * params.channel_count * params.spw_count,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.spw_index_array, sizeof(unsigned int) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.flagged_rows, sizeof(bool) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.field_array, sizeof(unsigned int) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.visibility_weights, sizeof(visibility_weights_base_type) * params.chunk_max_row_count * params.channel_count * params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.flags, sizeof(bool) * params.chunk_max_row_count * params.channel_count * params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.output_buffer, sizeof(std::complex<grid_base_type>) * params.nx * params.ny * params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size));
	cudaSafeCall(hipMemset(gpu_params.output_buffer,0,sizeof(std::complex<grid_base_type>) * params.nx * params.ny * params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size));
	size_t size_of_convolution_function = (params.conv_support * 2 + 1 + 2) * params.conv_oversample; //see algorithms/convolution_policies.h for the reason behind the padding
	convolution_base_type * coalesced_filter = new convolution_base_type[size_of_convolution_function]();
	#pragma omp parallel for
	for (size_t x = 0; x < size_of_convolution_function; ++x){
	  size_t cs = x / params.conv_oversample;
	  size_t co = x % params.conv_oversample;
	  size_t new_index = co * (params.conv_support*2 + 3) + cs;
	  size_t old_index = params.conv_oversample * cs + co;
	  coalesced_filter[new_index] = params.conv[old_index];
	}
	cudaSafeCall(hipMalloc((void**)&gpu_params.conv, sizeof(convolution_base_type) * size_of_convolution_function));
	cudaSafeCall(hipMemcpy(gpu_params.conv, coalesced_filter, sizeof(convolution_base_type) * size_of_convolution_function,hipMemcpyHostToDevice));
	delete [] coalesced_filter;
	cudaSafeCall(hipMalloc((void**)&gpu_params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1)));
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    }
    void releaseLibrary() {
      hipDeviceSynchronize();
      cudaSafeCall(hipFree(gpu_params.output_buffer));
      cudaSafeCall(hipFree(gpu_params.visibilities));
      cudaSafeCall(hipFree(gpu_params.uvw_coords));
      cudaSafeCall(hipFree(gpu_params.reference_wavelengths));
      cudaSafeCall(hipFree(gpu_params.enabled_channels));
      cudaSafeCall(hipFree(gpu_params.channel_grid_indicies));
      cudaSafeCall(hipFree(gpu_params.spw_index_array));
      cudaSafeCall(hipFree(gpu_params.flagged_rows));
      cudaSafeCall(hipFree(gpu_params.field_array));
      cudaSafeCall(hipFree(gpu_params.flags));
      cudaSafeCall(hipFree(gpu_params.conv));
      cudaSafeCall(hipFree(gpu_params.baseline_starting_indexes));
      cudaSafeCall(hipStreamDestroy(compute_stream));
      delete gridding_walltime;
      delete inversion_walltime;
      hipDeviceReset(); //leave the device in a safe state
    }
    void weight_uniformly(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: weight_uniformly");
    }
    void grid_single_pol(gridding_parameters & params){
      gridding_walltime->start();
      printf("Gridding single correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	size_t ubound = params.row_count*params.channel_count;
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	//Do not parallelize this:
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / params.channel_count;
	    size_t c = i - r * params.channel_count;
	    size_t compact_index = r*params.channel_count + c;
	    size_t strided_index = (compact_index)*params.number_of_polarization_terms + params.polarization_index;
	    params.visibilities[compact_index] = params.visibilities[strided_index];
	    params.visibility_weights[compact_index] = params.visibility_weights[strided_index];
	    params.flags[compact_index] = params.flags[strided_index];
	}
	
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
      }
      //invoke computation
      {
	size_t conv_support_size = (params.conv_support*2+1);
	size_t padded_conv_support_size = (conv_support_size+2);
	size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size;
	size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	
	dim3 no_threads_per_block(block_size,1,1);
	dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	typedef imaging::correlation_gridding_policy<imaging::grid_single_correlation> correlation_gridding_policy;
	imaging::templated_gridder<correlation_gridding_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
      }
      //swap buffers device -> host when gridded last chunk
      if (params.is_final_data_chunk){
	gridding_barrier();
	cudaSafeCall(hipMemcpy(params.output_buffer,gpu_params.output_buffer,sizeof(std::complex<grid_base_type>) * params.nx * params.ny * params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size,hipMemcpyDeviceToHost));
      }      
      gridding_walltime->stop();
    }
    void facet_single_pol(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_single_pol");
    }
    void grid_duel_pol(gridding_parameters & params){
      gridding_walltime->start();
      printf("Gridding duel correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count  * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	size_t ubound = params.row_count*params.channel_count;
	//Do not parallelize this:
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / (params.channel_count);
	    size_t c = i - r * params.channel_count;
	    size_t channel_flat_index = r*params.channel_count + c;
	    size_t strided_index_corr1 = (channel_flat_index)*params.number_of_polarization_terms + params.polarization_index;
	    size_t strided_index_corr2 = (channel_flat_index)*params.number_of_polarization_terms + params.second_polarization_index;
	    size_t compact_index_corr1 = channel_flat_index << 1;
	    size_t compact_index_corr2 = compact_index_corr1 + 1;
	    params.visibilities[compact_index_corr1] = params.visibilities[strided_index_corr1];
	    params.visibility_weights[compact_index_corr1] = params.visibility_weights[strided_index_corr1];
	    params.flags[compact_index_corr1] = params.flags[strided_index_corr1];
	    params.visibilities[compact_index_corr2] = params.visibilities[strided_index_corr2];
	    params.visibility_weights[compact_index_corr2] = params.visibility_weights[strided_index_corr2];
	    params.flags[compact_index_corr2] = params.flags[strided_index_corr2];
	}
	
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
	{
	  size_t conv_support_size = (params.conv_support*2+1);
	  size_t padded_conv_support_size = (conv_support_size+2);
	  size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size;
	  size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	  size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	  size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	  dim3 no_threads_per_block(block_size,1,1);
	  dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	  size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	  typedef imaging::correlation_gridding_policy<imaging::grid_duel_correlation> correlation_gridding_policy;
	  imaging::templated_gridder<correlation_gridding_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
	}
	//swap buffers device -> host when gridded last chunk
	if (params.is_final_data_chunk){
	  gridding_barrier();
	  cudaSafeCall(hipMemcpy(params.output_buffer,gpu_params.output_buffer,sizeof(std::complex<grid_base_type>) * params.nx * params.ny * 
				  params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size,hipMemcpyDeviceToHost));
	}      
      }
      gridding_walltime->stop();
    }
    void facet_duel_pol(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_duel_pol");
    }
    void grid_4_cor(gridding_parameters & params){
      gridding_walltime->start();
      printf("Gridding 4 correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//copy the read chunk accross to the GPU
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count  * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
	{
	  size_t conv_support_size = (params.conv_support*2+1);
	  size_t padded_conv_support_size = (conv_support_size+2);
	  size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size;
	  size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	  size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	  size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	  dim3 no_threads_per_block(block_size,1,1);
	  dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	  size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	  typedef imaging::correlation_gridding_policy<imaging::grid_4_correlation> correlation_gridding_policy;
	  imaging::templated_gridder<correlation_gridding_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
	}
	//swap buffers device -> host when gridded last chunk
	if (params.is_final_data_chunk){
	  gridding_barrier();
	  cudaSafeCall(hipMemcpy(params.output_buffer,gpu_params.output_buffer,sizeof(std::complex<grid_base_type>) * params.nx * params.ny * 
				  params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size,hipMemcpyDeviceToHost));
	}      
      }
      gridding_walltime->stop();
    }
    void facet_4_cor(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_4_cor");
    }
    void facet_4_cor_corrections(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_4_cor_corrections");
    }
    void grid_sampling_function(gridding_parameters & params){
//       throw std::runtime_error("Backend Unimplemented exception: grid_sampling_function");
    }
    void facet_sampling_function(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_sampling_function");
    }
}
