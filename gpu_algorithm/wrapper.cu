#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <numeric>
#include <cstring>

#include "gpu_wrapper.h"
#include "dft.h"
#include "templated_gridder.h"

#include "timer.h"
#include "correlation_gridding_traits.h"
#include "correlation_gridding_policies.h"
#include "baseline_transform_policies.h"
#include "phase_transform_policies.h"
#include "jones_2x2.h"

#define NO_THREADS_PER_BLOCK_DIM 256

extern "C" {
    utils::timer * gridding_walltime;
    hipStream_t compute_stream;
    gridding_parameters gpu_params;
    bool initialized = false;
    double get_gridding_walltime(){
      return gridding_walltime->duration();
    }
    void gridding_barrier(){
      cudaSafeCall(hipStreamSynchronize(compute_stream));
    }
    void initLibrary(gridding_parameters & params) {
	if (initialized) return;
	initialized = true;
        int num_devices, device;
        hipGetDeviceCount(&num_devices);
        if (num_devices > 0) {
            //get the argmax{devID}(multiProcessorCounts):
            int max_multiprocessors = 0, max_device = 0;
            for (device = 0; device < num_devices; device++) {
                hipDeviceProp_t properties;
                hipGetDeviceProperties(&properties, device);
                if (max_multiprocessors < properties.multiProcessorCount) {
                    max_multiprocessors = properties.multiProcessorCount;
                    max_device = device;
                }
            }
            hipSetDevice(max_device); //select device
            hipDeviceReset(); //ensure device is in a safe state before we begin processing

            //print some stats:
            hipDeviceProp_t properties;
            hipGetDeviceProperties(&properties, max_device);

            size_t mem_tot = 0;
            size_t mem_free = 0;
            hipMemGetInfo  (&mem_free, & mem_tot);
            printf("---------------------------------------Backend: GPU GRIDDING LIBRARY---------------------------------------\n");
            printf("%s, device %d on PCI Bus #%d, clocked at %f GHz\n",properties.name,properties.pciDeviceID,
                   properties.pciBusID,properties.clockRate / 1000000.0);
            printf("Compute capability %d.%d with %f GiB global memory (%f GiB free)\n",properties.major,
                   properties.minor,mem_tot/1024.0/1024.0/1024.0,mem_free/1024.0/1024.0/1024.0);
            printf("%d SMs are available\n",properties.multiProcessorCount);
            printf("-----------------------------------------------------------------------------------------------------------\n");
        } else 
            throw std::runtime_error("Cannot find suitable GPU device. Giving up");
	cudaSafeCall(hipStreamCreateWithFlags(&compute_stream,hipStreamNonBlocking));
	gridding_walltime = new utils::timer(compute_stream);
	//alloc memory for all the arrays on the gpu at the beginning of execution...
	gpu_params = params;
	cudaSafeCall(hipMalloc((void**)&gpu_params.visibilities, sizeof(std::complex<visibility_base_type>) * params.chunk_max_row_count*params.channel_count*params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.uvw_coords, sizeof(imaging::uvw_coord<uvw_base_type>) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.reference_wavelengths, sizeof(reference_wavelengths_base_type) * params.channel_count * params.spw_count));
	cudaSafeCall(hipMemcpy(gpu_params.reference_wavelengths,params.reference_wavelengths,sizeof(reference_wavelengths_base_type) * params.channel_count * params.spw_count,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.enabled_channels, sizeof(bool) * params.channel_count * params.spw_count));
	cudaSafeCall(hipMemcpy(gpu_params.enabled_channels,params.enabled_channels, sizeof(bool) * params.channel_count * params.spw_count,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.channel_grid_indicies, sizeof(size_t) * params.channel_count * params.spw_count));
	cudaSafeCall(hipMemcpy(gpu_params.channel_grid_indicies,params.channel_grid_indicies, sizeof(size_t) * params.channel_count * params.spw_count,hipMemcpyHostToDevice));
	if (params.should_grid_sampling_function){
	  cudaSafeCall(hipMalloc((void**)&gpu_params.sampling_function_channel_grid_indicies, sizeof(size_t) * params.channel_count * params.spw_count))
	  cudaSafeCall(hipMemcpy(gpu_params.sampling_function_channel_grid_indicies,
				  params.sampling_function_channel_grid_indicies, 
				  sizeof(size_t) * params.channel_count * params.spw_count,hipMemcpyHostToDevice));
	}
	cudaSafeCall(hipMalloc((void**)&gpu_params.spw_index_array, sizeof(unsigned int) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.flagged_rows, sizeof(bool) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.field_array, sizeof(unsigned int) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.visibility_weights, sizeof(visibility_weights_base_type) * params.chunk_max_row_count * params.channel_count * params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.flags, sizeof(bool) * params.chunk_max_row_count * params.channel_count * params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.output_buffer, sizeof(std::complex<grid_base_type>) * params.nx * params.ny * 
								    params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size * params.num_facet_centres));
	cudaSafeCall(hipMemcpy(gpu_params.output_buffer,params.output_buffer,sizeof(std::complex<grid_base_type>) * params.nx * params.ny * 
				params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size * params.num_facet_centres,hipMemcpyHostToDevice));
	if (params.should_grid_sampling_function){
	  cudaSafeCall(hipMalloc((void**)&gpu_params.sampling_function_buffer, sizeof(std::complex<grid_base_type>) * params.nx * params.ny * 
							params.sampling_function_channel_count * params.num_facet_centres));
	  cudaSafeCall(hipMemcpy(gpu_params.sampling_function_buffer,params.sampling_function_buffer,sizeof(std::complex<grid_base_type>) * params.nx * params.ny * 
						    params.sampling_function_channel_count * params.num_facet_centres,hipMemcpyHostToDevice));
	}
	size_t size_of_convolution_function = (params.conv_support * 2 + 1 + 2) * params.conv_oversample; //see algorithms/convolution_policies.h for the reason behind the padding
	cudaSafeCall(hipMalloc((void**)&gpu_params.conv, sizeof(convolution_base_type) * size_of_convolution_function));
	cudaSafeCall(hipMemcpy(gpu_params.conv, params.conv, sizeof(convolution_base_type) * size_of_convolution_function,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1)));
	cudaSafeCall(hipMalloc((void**)&gpu_params.facet_centres, sizeof(uvw_base_type) * params.num_facet_centres * 2)); //enough space to store ra,dec coordinates of facet delay centres
	cudaSafeCall(hipMemcpy(gpu_params.facet_centres,params.facet_centres, sizeof(uvw_base_type) * params.num_facet_centres * 2,hipMemcpyHostToDevice));
	if (gpu_params.should_invert_jones_terms){
	  cudaSafeCall(hipMalloc((void**)&gpu_params.antenna_1_ids, sizeof(unsigned int) * (params.chunk_max_row_count)));
	  cudaSafeCall(hipMalloc((void**)&gpu_params.antenna_2_ids, sizeof(unsigned int) * (params.chunk_max_row_count)));
	  cudaSafeCall(hipMalloc((void**)&gpu_params.timestamp_ids, sizeof(size_t) * (params.chunk_max_row_count)));
	  size_t no_timesteps_needed = (params.chunk_max_row_count / params.baseline_count + 1);
	  cudaSafeCall(hipMalloc((void**)&gpu_params.jones_terms,sizeof(imaging::jones_2x2<visibility_base_type>) * (no_timesteps_needed *
												  params.antenna_count *
												  params.num_facet_centres * 
												  params.spw_count * 
												  params.channel_count)));
	  cudaSafeCall(hipMalloc((void**)&gpu_params.antenna_jones_starting_indexes,sizeof(size_t) * (params.antenna_count + 1)));
	  cudaSafeCall(hipMalloc((void**)&gpu_params.jones_time_indicies_per_antenna,sizeof(size_t) * params.antenna_count * no_timesteps_needed));
	}
	cudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    }
    void releaseLibrary() {
      if (!initialized) return;
      initialized = false;
      hipDeviceSynchronize();
      cudaSafeCall(hipFree(gpu_params.output_buffer));
      if (gpu_params.should_grid_sampling_function)
	cudaSafeCall(hipFree(gpu_params.sampling_function_buffer));
      cudaSafeCall(hipFree(gpu_params.visibilities));
      cudaSafeCall(hipFree(gpu_params.uvw_coords));
      cudaSafeCall(hipFree(gpu_params.reference_wavelengths));
      cudaSafeCall(hipFree(gpu_params.enabled_channels));
      cudaSafeCall(hipFree(gpu_params.channel_grid_indicies));
      if (gpu_params.should_grid_sampling_function)
	cudaSafeCall(hipFree(gpu_params.sampling_function_channel_grid_indicies));
      cudaSafeCall(hipFree(gpu_params.spw_index_array));
      cudaSafeCall(hipFree(gpu_params.flagged_rows));
      cudaSafeCall(hipFree(gpu_params.field_array));
      cudaSafeCall(hipFree(gpu_params.flags));
      cudaSafeCall(hipFree(gpu_params.conv));
      cudaSafeCall(hipFree(gpu_params.baseline_starting_indexes));
      cudaSafeCall(hipFree(gpu_params.facet_centres));
      if (gpu_params.should_invert_jones_terms){
	cudaSafeCall(hipFree(gpu_params.antenna_1_ids));
	cudaSafeCall(hipFree(gpu_params.antenna_2_ids));
	cudaSafeCall(hipFree(gpu_params.timestamp_ids));
	cudaSafeCall(hipFree(gpu_params.jones_terms));
	cudaSafeCall(hipFree(gpu_params.antenna_jones_starting_indexes));
	cudaSafeCall(hipFree(gpu_params.jones_time_indicies_per_antenna));
      }
      cudaSafeCall(hipStreamDestroy(compute_stream));
      delete gridding_walltime;
      hipDeviceReset(); //leave the device in a safe state
    }
    void weight_uniformly(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: weight_uniformly");
    }
    void copy_back_grid_if_last_stamp(gridding_parameters & params,const gridding_parameters & gpu_params){
      if (params.is_final_data_chunk){
	gridding_barrier();
	cudaSafeCall(hipMemcpy(params.output_buffer,gpu_params.output_buffer,sizeof(std::complex<grid_base_type>) * params.nx * params.ny * 
				params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size * params.num_facet_centres,
				hipMemcpyDeviceToHost));
      }  
    }
    void copy_back_sampling_function_if_last_stamp(gridding_parameters & params,const gridding_parameters & gpu_params){
      if (params.is_final_data_chunk){
	gridding_barrier();
	cudaSafeCall(hipMemcpy(params.sampling_function_buffer,gpu_params.sampling_function_buffer,sizeof(std::complex<grid_base_type>) * params.nx * params.ny * 
				params.sampling_function_channel_count * params.num_facet_centres,
				hipMemcpyDeviceToHost));
      }  
    }
    void grid_single_pol(gridding_parameters & params){
      gridding_walltime->start();
      printf("Gridding single correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	size_t ubound = params.row_count*params.channel_count;
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	//Do not parallelize this:
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / params.channel_count;
	    size_t c = i - r * params.channel_count;
	    size_t compact_index = r*params.channel_count + c;
	    size_t strided_index = (compact_index)*params.number_of_polarization_terms + params.polarization_index;
	    params.visibilities[compact_index] = params.visibilities[strided_index];
	    params.visibility_weights[compact_index] = params.visibility_weights[strided_index];
	    params.flags[compact_index] = params.flags[strided_index];
	}
	
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
      }
      //invoke computation
      {
	size_t conv_support_size = (params.conv_support*2+1);
	size_t padded_conv_support_size = (conv_support_size+2);
	size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size;
	size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	
	dim3 no_threads_per_block(block_size,1,1);
	dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	typedef imaging::correlation_gridding_policy<imaging::grid_single_correlation> correlation_gridding_policy;
	typedef imaging::baseline_transform_policy<imaging::transform_disable_facet_rotation > baseline_transform_policy;
	typedef imaging::phase_transform_policy<imaging::disable_faceting_phase_shift> phase_transform_policy;
	imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
      }
      //swap buffers device -> host when gridded last chunk
      copy_back_grid_if_last_stamp(params,gpu_params);
      gridding_walltime->stop();
    }
    void facet_single_pol(gridding_parameters & params){
     gridding_walltime->start();
      printf("Faceting single correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	size_t ubound = params.row_count*params.channel_count;
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));

	//Do not parallelize this:
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / params.channel_count;
	    size_t c = i - r * params.channel_count;
	    size_t compact_index = r*params.channel_count + c;
	    size_t strided_index = (compact_index)*params.number_of_polarization_terms + params.polarization_index;
	    params.visibilities[compact_index] = params.visibilities[strided_index];
	    params.visibility_weights[compact_index] = params.visibility_weights[strided_index];
	    params.flags[compact_index] = params.flags[strided_index];
	}
	
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
      }
      //invoke computation
      {
	size_t conv_support_size = (params.conv_support*2+1);
	size_t padded_conv_support_size = (conv_support_size+2);
	size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size * params.num_facet_centres;
	size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	
	dim3 no_threads_per_block(block_size,1,1);
	dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	typedef imaging::correlation_gridding_policy<imaging::grid_single_correlation> correlation_gridding_policy;
	typedef imaging::baseline_transform_policy<imaging::transform_facet_lefthanded_ra_dec > baseline_transform_policy;
	typedef imaging::phase_transform_policy<imaging::enable_faceting_phase_shift> phase_transform_policy;
	imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
      }
      //swap buffers device -> host when gridded last chunk
      copy_back_grid_if_last_stamp(params,gpu_params);
      gridding_walltime->stop();
    }
    void grid_duel_pol(gridding_parameters & params){
      gridding_walltime->start();
      printf("Gridding duel correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count  * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	size_t ubound = params.row_count*params.channel_count;
	//Do not parallelize this:
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / (params.channel_count);
	    size_t c = i - r * params.channel_count;
	    size_t channel_flat_index = r*params.channel_count + c;
	    size_t strided_index_corr1 = (channel_flat_index)*params.number_of_polarization_terms + params.polarization_index;
	    size_t strided_index_corr2 = (channel_flat_index)*params.number_of_polarization_terms + params.second_polarization_index;
	    size_t compact_index_corr1 = channel_flat_index << 1;
	    size_t compact_index_corr2 = compact_index_corr1 + 1;
	    params.visibilities[compact_index_corr1] = params.visibilities[strided_index_corr1];
	    params.visibility_weights[compact_index_corr1] = params.visibility_weights[strided_index_corr1];
	    params.flags[compact_index_corr1] = params.flags[strided_index_corr1];
	    params.visibilities[compact_index_corr2] = params.visibilities[strided_index_corr2];
	    params.visibility_weights[compact_index_corr2] = params.visibility_weights[strided_index_corr2];
	    params.flags[compact_index_corr2] = params.flags[strided_index_corr2];
	}
	
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
	{
	  size_t conv_support_size = (params.conv_support*2+1);
	  size_t padded_conv_support_size = (conv_support_size+2);
	  size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size;
	  size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	  size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	  size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	  dim3 no_threads_per_block(block_size,1,1);
	  dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	  size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	  typedef imaging::correlation_gridding_policy<imaging::grid_duel_correlation> correlation_gridding_policy;
	  typedef imaging::baseline_transform_policy<imaging::transform_disable_facet_rotation > baseline_transform_policy;
	  typedef imaging::phase_transform_policy<imaging::disable_faceting_phase_shift> phase_transform_policy;
	  imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
	}
	//swap buffers device -> host when gridded last chunk
	copy_back_grid_if_last_stamp(params,gpu_params);    
      }
      gridding_walltime->stop();
    }
    void facet_duel_pol(gridding_parameters & params){
      gridding_walltime->start();
      printf("Faceting duel correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count  * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	size_t ubound = params.row_count*params.channel_count;
	//Do not parallelize this:
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / (params.channel_count);
	    size_t c = i - r * params.channel_count;
	    size_t channel_flat_index = r*params.channel_count + c;
	    size_t strided_index_corr1 = (channel_flat_index)*params.number_of_polarization_terms + params.polarization_index;
	    size_t strided_index_corr2 = (channel_flat_index)*params.number_of_polarization_terms + params.second_polarization_index;
	    size_t compact_index_corr1 = channel_flat_index << 1;
	    size_t compact_index_corr2 = compact_index_corr1 + 1;
	    params.visibilities[compact_index_corr1] = params.visibilities[strided_index_corr1];
	    params.visibility_weights[compact_index_corr1] = params.visibility_weights[strided_index_corr1];
	    params.flags[compact_index_corr1] = params.flags[strided_index_corr1];
	    params.visibilities[compact_index_corr2] = params.visibilities[strided_index_corr2];
	    params.visibility_weights[compact_index_corr2] = params.visibility_weights[strided_index_corr2];
	    params.flags[compact_index_corr2] = params.flags[strided_index_corr2];
	}
	
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
	{
	  size_t conv_support_size = (params.conv_support*2+1);
	  size_t padded_conv_support_size = (conv_support_size+2);
	  size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size * params.num_facet_centres;
	  size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	  size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	  size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	  dim3 no_threads_per_block(block_size,1,1);
	  dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	  size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	  typedef imaging::correlation_gridding_policy<imaging::grid_duel_correlation> correlation_gridding_policy;
	  typedef imaging::baseline_transform_policy<imaging::transform_facet_lefthanded_ra_dec > baseline_transform_policy;
	  typedef imaging::phase_transform_policy<imaging::enable_faceting_phase_shift> phase_transform_policy;
	  imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
	}
	//swap buffers device -> host when gridded last chunk
	copy_back_grid_if_last_stamp(params,gpu_params);    
      }
      gridding_walltime->stop();
    }
    void grid_4_cor(gridding_parameters & params){
      gridding_walltime->start();
      printf("Gridding 4 correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//copy the read chunk accross to the GPU
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count  * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
	{
	  size_t conv_support_size = (params.conv_support*2+1);
	  size_t padded_conv_support_size = (conv_support_size+2);
	  size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size;
	  size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	  size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	  size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	  dim3 no_threads_per_block(block_size,1,1);
	  dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	  size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	  typedef imaging::correlation_gridding_policy<imaging::grid_4_correlation> correlation_gridding_policy;
	  typedef imaging::baseline_transform_policy<imaging::transform_disable_facet_rotation > baseline_transform_policy;
	  typedef imaging::phase_transform_policy<imaging::disable_faceting_phase_shift> phase_transform_policy;
	  imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
	}
	//swap buffers device -> host when gridded last chunk
	copy_back_grid_if_last_stamp(params,gpu_params);    
      }
      gridding_walltime->stop();
    }
    void facet_4_cor(gridding_parameters & params){
      gridding_walltime->start();
      printf("Faceting 4 correlation on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//copy the read chunk accross to the GPU
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count  * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
	{
	  size_t conv_support_size = (params.conv_support*2+1);
	  size_t padded_conv_support_size = (conv_support_size+2);
	  size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size * params.num_facet_centres;
	  size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	  size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	  size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	  dim3 no_threads_per_block(block_size,1,1);
	  dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	  size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	  typedef imaging::correlation_gridding_policy<imaging::grid_4_correlation> correlation_gridding_policy;
	  typedef imaging::baseline_transform_policy<imaging::transform_facet_lefthanded_ra_dec > baseline_transform_policy;
	  typedef imaging::phase_transform_policy<imaging::enable_faceting_phase_shift> phase_transform_policy;
	  imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
	}
	//swap buffers device -> host when gridded last chunk
	copy_back_grid_if_last_stamp(params,gpu_params);    
      }
      gridding_walltime->stop();
    }
    void facet_4_cor_corrections(gridding_parameters & params){
      gridding_walltime->start();
      printf("Faceting 4 correlation on the GPU with Jones corrections...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//copy the read chunk accross to the GPU
	cudaSafeCall(hipHostRegister(params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count  * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipHostRegister(params.antenna_1_ids, sizeof(unsigned int) * (params.row_count), 0));
	cudaSafeCall(hipHostRegister(params.antenna_2_ids, sizeof(unsigned int) * (params.row_count), 0));
	cudaSafeCall(hipHostRegister(params.timestamp_ids, sizeof(std::size_t) * (params.row_count), 0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count * params.number_of_polarization_terms_being_gridded,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.antenna_1_ids,params.antenna_1_ids,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.antenna_2_ids,params.antenna_2_ids,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.timestamp_ids,params.timestamp_ids,sizeof(std::size_t) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	//repack the jones terms per antenna so that we don't transfer unnecessary stuff over PCI-e
	{
	  using namespace std;
	  //we want to retrieve the # timestamps for every antenna that means we need (n+1) elements in this array
	  vector<size_t> antenna_timestamp_starting_indexes(params.antenna_count + 1,0);
	  //bin
	  {
	    vector<long> antenna_current_timestamp(params.antenna_count,-1);
	    for(size_t row = 0; row < params.row_count; ++row){
	      if ((long)params.timestamp_ids[row] > antenna_current_timestamp[params.antenna_1_ids[row]]){
		antenna_timestamp_starting_indexes[params.antenna_1_ids[row] + 1] += 1;
		antenna_current_timestamp[params.antenna_1_ids[row]] += 1;
	      }
	      if ((long)params.timestamp_ids[row] > antenna_current_timestamp[params.antenna_2_ids[row]]){
		antenna_timestamp_starting_indexes[params.antenna_2_ids[row] + 1] += 1;
		antenna_current_timestamp[params.antenna_2_ids[row]] += 1;
	      }
	    }
	  }
	  //because we binned at antenna_id + 1 partial_sum will compute the prescan (starting timestamp index per antenna)
	  std::partial_sum(antenna_timestamp_starting_indexes.begin(),
			   antenna_timestamp_starting_indexes.end(),
			   antenna_timestamp_starting_indexes.begin());
	  size_t step_size = params.num_facet_centres * params.spw_count * params.channel_count;
	  vector<imaging::jones_2x2<visibility_base_type> > repacked_data(antenna_timestamp_starting_indexes[(params.antenna_count)] * step_size);
	  vector<std::size_t> repacked_indexes(antenna_timestamp_starting_indexes[(params.antenna_count)]);
	  cudaSafeCall(hipHostRegister(&repacked_data[0], sizeof(imaging::jones_2x2<visibility_base_type>) * repacked_data.size(), 0));
	  cudaSafeCall(hipHostRegister(&antenna_timestamp_starting_indexes[0], sizeof(size_t) * antenna_timestamp_starting_indexes.size(), 0));
	  cudaSafeCall(hipMemcpyAsync(gpu_params.antenna_jones_starting_indexes,&antenna_timestamp_starting_indexes[0],
				       sizeof(size_t) * antenna_timestamp_starting_indexes.size(),hipMemcpyHostToDevice,compute_stream));
	  cudaSafeCall(hipHostRegister(&repacked_indexes[0], sizeof(size_t) * repacked_indexes.size(), 0));
	  { //now repack
	    vector<long> antenna_current_timestamp(params.antenna_count,-1);

	    for(size_t row = 0; row < params.row_count; ++row){
	      //copy the first antenna into position
	      if ((long)params.timestamp_ids[row] > antenna_current_timestamp[params.antenna_1_ids[row]]){ //a single antenna may be in multiple baselines... don't recopy
		antenna_current_timestamp[params.antenna_1_ids[row]] += 1;
		size_t old_index_antenna_1 = (params.timestamp_ids[row] * params.antenna_count + params.antenna_1_ids[row]) * 
					      step_size;
		size_t new_index_antenna_1 = (antenna_timestamp_starting_indexes[params.antenna_1_ids[row]] +
					      antenna_current_timestamp[params.antenna_1_ids[row]]) *
					      step_size;
		repacked_indexes[new_index_antenna_1/step_size] = params.timestamp_ids[row];
		imaging::jones_2x2<visibility_base_type> * old_arr = (imaging::jones_2x2<visibility_base_type> *) params.jones_terms;
		memcpy(&repacked_data[0] + new_index_antenna_1,
		       old_arr + old_index_antenna_1,
		       step_size * sizeof(imaging::jones_2x2<visibility_base_type>));
	      }
	      //copy the second antenna into position
	      if ((long)params.timestamp_ids[row] > antenna_current_timestamp[params.antenna_2_ids[row]]){ //a single antenna may be in multiple baselines... don't recopy
		antenna_current_timestamp[params.antenna_2_ids[row]] += 1;
		size_t old_index_antenna_2 = (params.timestamp_ids[row] * params.antenna_count + params.antenna_2_ids[row]) * 
					      step_size;
		size_t new_index_antenna_2 = (antenna_timestamp_starting_indexes[params.antenna_2_ids[row]] +
					      antenna_current_timestamp[params.antenna_2_ids[row]]) *
					      step_size;
		repacked_indexes[new_index_antenna_2/step_size] = params.timestamp_ids[row];
		imaging::jones_2x2<visibility_base_type> * old_arr = (imaging::jones_2x2<visibility_base_type> *) params.jones_terms;
		memcpy(&repacked_data[0] + new_index_antenna_2,
		       old_arr + old_index_antenna_2,
		       step_size * sizeof(imaging::jones_2x2<visibility_base_type>));
	      }
	    }
	  }
	  printf("INVERTING %lu JONES MATRICIES\n",repacked_data.size());
	  invert_all(&repacked_data[0], repacked_data.size());
	  cudaSafeCall(hipMemcpyAsync(gpu_params.jones_terms,&repacked_data[0],sizeof(imaging::jones_2x2<visibility_base_type>) * repacked_data.size(),
		       hipMemcpyHostToDevice,compute_stream));
	  cudaSafeCall(hipMemcpyAsync(gpu_params.jones_time_indicies_per_antenna,&repacked_indexes[0],sizeof(size_t) * repacked_indexes.size(),
		       hipMemcpyHostToDevice,compute_stream));
	  cudaSafeCall(hipHostUnregister(&repacked_data[0]));
	  cudaSafeCall(hipHostUnregister(&antenna_timestamp_starting_indexes[0]));
	  cudaSafeCall(hipHostUnregister(&repacked_indexes[0]));
	  printf("TRANSFERRED %lu REPACKED JONES MATRICIES TO DEVICE\n",repacked_data.size());
	}
	
	cudaSafeCall(hipHostUnregister(params.visibilities));
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.visibility_weights));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
	cudaSafeCall(hipHostUnregister(params.antenna_1_ids));
	cudaSafeCall(hipHostUnregister(params.antenna_2_ids));
	cudaSafeCall(hipHostUnregister(params.timestamp_ids));
	{
	  size_t conv_support_size = (params.conv_support*2+1);
	  size_t padded_conv_support_size = (conv_support_size+2);
	  size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size * params.num_facet_centres;
	  size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	  size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	  size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	  dim3 no_threads_per_block(block_size,1,1);
	  dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	  size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	  typedef imaging::correlation_gridding_policy<imaging::grid_4_correlation> correlation_gridding_policy;
	  typedef imaging::baseline_transform_policy<imaging::transform_facet_lefthanded_ra_dec > baseline_transform_policy;
	  typedef imaging::phase_transform_policy<imaging::enable_faceting_phase_shift> phase_transform_policy;
	  imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
	}
	//swap buffers device -> host when gridded last chunk
	copy_back_grid_if_last_stamp(params,gpu_params);    
      }
      gridding_walltime->stop();
    }
    void grid_sampling_function(gridding_parameters & params){
      gridding_walltime->start();
      printf("Gridding sampling function on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	size_t ubound = params.row_count*params.channel_count;
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	//Do not parallelize this:
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / params.channel_count;
	    size_t c = i - r * params.channel_count;
	    size_t compact_index = r*params.channel_count + c;
	    size_t strided_index = (compact_index)*params.number_of_polarization_terms + params.polarization_index;
	    params.flags[compact_index] = params.flags[strided_index];
	}
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
      }
      //invoke computation
      {
	size_t conv_support_size = (params.conv_support*2+1);
	size_t padded_conv_support_size = (conv_support_size+2);
	size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size;
	size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	
	dim3 no_threads_per_block(block_size,1,1);
	dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	typedef imaging::correlation_gridding_policy<imaging::grid_sampling_function> correlation_gridding_policy;
	typedef imaging::baseline_transform_policy<imaging::transform_disable_facet_rotation > baseline_transform_policy;
	typedef imaging::phase_transform_policy<imaging::disable_faceting_phase_shift> phase_transform_policy;
	imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
      }
      //swap buffers device -> host when gridded last chunk
      copy_back_sampling_function_if_last_stamp(params,gpu_params);
      gridding_walltime->stop();
    }
    void facet_sampling_function(gridding_parameters & params){
      gridding_walltime->start();
      printf("Faceting sampling function on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	gpu_params.is_final_data_chunk = params.is_final_data_chunk;
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	size_t ubound = params.row_count*params.channel_count;
	cudaSafeCall(hipHostRegister(params.spw_index_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flagged_rows,sizeof(bool) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.flags,sizeof(bool) * params.row_count * params.channel_count,0));
	cudaSafeCall(hipHostRegister(params.field_array,sizeof(unsigned int) * params.row_count,0));
	cudaSafeCall(hipHostRegister(params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),0));
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.baseline_starting_indexes, params.baseline_starting_indexes, sizeof(size_t) * (params.baseline_count+1),hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.field_array,params.field_array,sizeof(unsigned int) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	//Do not parallelize this:
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / params.channel_count;
	    size_t c = i - r * params.channel_count;
	    size_t compact_index = r*params.channel_count + c;
	    size_t strided_index = (compact_index)*params.number_of_polarization_terms + params.polarization_index;
	    params.flags[compact_index] = params.flags[strided_index];
	}
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	
	cudaSafeCall(hipHostUnregister(params.spw_index_array));
	cudaSafeCall(hipHostUnregister(params.uvw_coords));
	cudaSafeCall(hipHostUnregister(params.flagged_rows));
	cudaSafeCall(hipHostUnregister(params.flags));
	cudaSafeCall(hipHostUnregister(params.field_array));
	cudaSafeCall(hipHostUnregister(params.baseline_starting_indexes));
      }
      //invoke computation
      {
	size_t conv_support_size = (params.conv_support*2+1);
	size_t padded_conv_support_size = (conv_support_size+2);
	size_t min_threads_needed = params.baseline_count * conv_support_size * conv_support_size * params.num_facet_centres;
	size_t block_size = NO_THREADS_PER_BLOCK_DIM;
	size_t total_blocks_needed = ceil(min_threads_needed / double(block_size));
	size_t total_blocks_needed_per_dim = total_blocks_needed;
	
	
	dim3 no_threads_per_block(block_size,1,1);
	dim3 no_blocks_per_grid(total_blocks_needed_per_dim,1,1);
	size_t size_of_convolution_function = padded_conv_support_size * params.conv_oversample * sizeof(convolution_base_type); //see algorithms/convolution_policies.h for the reason behind the padding
	typedef imaging::correlation_gridding_policy<imaging::grid_sampling_function> correlation_gridding_policy;
	typedef imaging::baseline_transform_policy<imaging::transform_facet_lefthanded_ra_dec > baseline_transform_policy;
	typedef imaging::phase_transform_policy<imaging::enable_faceting_phase_shift> phase_transform_policy;
	imaging::templated_gridder<correlation_gridding_policy,baseline_transform_policy,phase_transform_policy><<<no_blocks_per_grid,no_threads_per_block,size_of_convolution_function,compute_stream>>>(gpu_params);
      }
      //swap buffers device -> host when gridded last chunk
      copy_back_sampling_function_if_last_stamp(params,gpu_params);
      gridding_walltime->stop();
    }
}
