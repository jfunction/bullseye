#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cstdio>
#include "wrapper.h"
#include "dft.h"
#include "gridder.h"
#include "timer.h"
#include "cu_common.h"
#include "uvw_coord.h"
#include "fft_shift_utils.h"
#define NO_THREADS_PER_BLOCK_DIM 16

extern imaging::uvw_coord< double > uvw;
extern "C" {
    utils::timer * gridding_walltime;
    utils::timer * inversion_walltime;
    hipStream_t compute_stream;
    
    gridding_parameters gpu_params;
    
    double get_gridding_walltime(){
      return gridding_walltime->duration();
    }
    double get_inversion_walltime(){
      return inversion_walltime->duration();
    }
    void gridding_barrier(){
      cudaSafeCall(hipStreamSynchronize(compute_stream));
    }
    void initLibrary(gridding_parameters & params) {
        int num_devices, device;
        hipGetDeviceCount(&num_devices);
        if (num_devices > 0) {
            //get the argmax{devID}(multiProcessorCounts):
            int max_multiprocessors = 0, max_device = 0;
            for (device = 0; device < num_devices; device++) {
                hipDeviceProp_t properties;
                hipGetDeviceProperties(&properties, device);
                if (max_multiprocessors < properties.multiProcessorCount) {
                    max_multiprocessors = properties.multiProcessorCount;
                    max_device = device;
                }
            }
            hipSetDevice(max_device); //select device
            hipDeviceReset(); //ensure device is in a safe state before we begin processing

            //print some stats:
            hipDeviceProp_t properties;
            hipGetDeviceProperties(&properties, max_device);

            size_t mem_tot = 0;
            size_t mem_free = 0;
            hipMemGetInfo  (&mem_free, & mem_tot);
            printf("-----------------------------------Backend: GPU DFT Library---------------------------------------\n");
            printf("%s, device %d on PCI Bus #%d, clocked at %f GHz\n",properties.name,properties.pciDeviceID,
                   properties.pciBusID,properties.clockRate / 1000000.0);
            printf("Compute capability %d.%d with %f GiB global memory (%f GiB free)\n",properties.major,
                   properties.minor,mem_tot/1024.0/1024.0/1024.0,mem_free/1024.0/1024.0/1024.0);
            printf("%d SMs are available\n",properties.multiProcessorCount);
            printf("--------------------------------------------------------------------------------------------------\n");
        } else 
            throw std::runtime_error("Cannot find suitable GPU device. Giving up");
	cudaSafeCall(hipStreamCreateWithFlags(&compute_stream,hipStreamNonBlocking));
	gridding_walltime = new utils::timer(compute_stream);
	inversion_walltime = new utils::timer(compute_stream);
	//alloc memory for all the arrays on the gpu at the beginning of execution...
	gpu_params = params;
	cudaSafeCall(hipMalloc((void**)&gpu_params.visibilities, sizeof(std::complex<visibility_base_type>) * params.chunk_max_row_count*params.channel_count*params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.uvw_coords, sizeof(imaging::uvw_coord<uvw_base_type>) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.reference_wavelengths, sizeof(reference_wavelengths_base_type) * params.channel_count));
	cudaSafeCall(hipMemcpy(gpu_params.reference_wavelengths,params.reference_wavelengths,sizeof(reference_wavelengths_base_type) * params.channel_count,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.enabled_channels, sizeof(bool) * params.channel_count));
	cudaSafeCall(hipMemcpy(gpu_params.enabled_channels,params.enabled_channels, sizeof(bool) * params.channel_count,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.channel_grid_indicies, sizeof(size_t) * params.channel_count));
	cudaSafeCall(hipMemcpy(gpu_params.channel_grid_indicies,params.channel_grid_indicies, sizeof(size_t) * params.channel_count,hipMemcpyHostToDevice));
	cudaSafeCall(hipMalloc((void**)&gpu_params.spw_index_array, sizeof(unsigned int) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.flagged_rows, sizeof(bool) * params.chunk_max_row_count));
	cudaSafeCall(hipMalloc((void**)&gpu_params.visibility_weights, sizeof(visibility_weights_base_type) * params.chunk_max_row_count * params.channel_count * params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.flags, sizeof(bool) * params.chunk_max_row_count * params.channel_count * params.number_of_polarization_terms_being_gridded));
	cudaSafeCall(hipMalloc((void**)&gpu_params.output_buffer, sizeof(std::complex<grid_base_type>) * params.nx * params.ny * params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size));
	cudaSafeCall(hipMemset(gpu_params.output_buffer,0,sizeof(grid_base_type) * params.nx * params.ny * params.number_of_polarization_terms_being_gridded * params.cube_channel_dim_size));
	cudaSafeCall(hipMalloc((void**)&gpu_params.conv, sizeof(convolution_base_type) * params.conv_support * params.conv_oversample));	
	cudaSafeCall(hipMemcpy(gpu_params.conv, params.conv, sizeof(convolution_base_type) * params.conv_support * params.conv_oversample,hipMemcpyHostToDevice));
	
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    }
    void releaseLibrary() {
      hipDeviceSynchronize();
      cudaSafeCall(hipFree(gpu_params.output_buffer));
      cudaSafeCall(hipFree(gpu_params.visibilities));
      cudaSafeCall(hipFree(gpu_params.uvw_coords));
      cudaSafeCall(hipFree(gpu_params.reference_wavelengths));
      cudaSafeCall(hipFree(gpu_params.enabled_channels));
      cudaSafeCall(hipFree(gpu_params.channel_grid_indicies));
      cudaSafeCall(hipFree(gpu_params.spw_index_array));
      cudaSafeCall(hipFree(gpu_params.flagged_rows));
      cudaSafeCall(hipFree(gpu_params.flags));
      cudaSafeCall(hipFree(gpu_params.conv));
      cudaSafeCall(hipStreamDestroy(compute_stream));
      hipDeviceReset(); //leave the device in a safe state
    }
    void weight_uniformly(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: weight_uniformly");
    }
    void finalize(gridding_parameters & params){
      gridding_barrier();
      inversion_walltime->start();
      std::size_t offset = params.nx*params.ny*params.cube_channel_dim_size*params.number_of_polarization_terms_being_gridded;
      /*
       * We'll be storing 32 bit real fits files so ignore all the imaginary components and cast whatever the grid was to float32
       */
      {
	  grid_base_type * __restrict__ grid_ptr_gridtype = (grid_base_type *)params.output_buffer;
	  float * __restrict__ grid_ptr_single = (float *)params.output_buffer;
	  for (std::size_t f = 0; f < params.num_facet_centres; ++f) {
	      std::size_t casting_lbound = offset*f;
	      std::size_t casting_ubound = casting_lbound + params.nx*params.ny*params.cube_channel_dim_size;
	      for (std::size_t i = casting_lbound; i < casting_ubound; ++i){
		  grid_ptr_single[i] = (float)(grid_ptr_gridtype[i]); //extract all the reals
	      }
	  }
      }
      inversion_walltime->stop();
    }
    void finalize_psf(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: finalize_psf");
    }
    void grid_single_pol(gridding_parameters & params){
      gridding_barrier();
      gridding_walltime->start();
      printf("Gridding single polarization on the GPU...\n");    
      //copy everything that changed to the gpu
      {
	gpu_params.row_count = params.row_count;
	gpu_params.no_timestamps_read = params.no_timestamps_read;
	cudaSafeCall(hipMemcpyAsync(gpu_params.uvw_coords,params.uvw_coords,sizeof(imaging::uvw_coord<uvw_base_type>) * params.row_count,hipMemcpyHostToDevice,compute_stream));
	//pack and cpy only the necessary visibilities (it doesn't matter if we mod the array here it is not being used again afterwards
	size_t ubound = params.row_count*params.channel_count;
	for (std::size_t i = 0; i < ubound; ++i){
	    size_t r = i / params.channel_count;
	    size_t c = i % params.channel_count;
	    size_t compact_index = r*params.channel_count + c;
	    size_t strided_index = (r*params.number_of_polarization_terms + params.polarization_index)*params.channel_count + c;
	    
	    params.visibilities[compact_index] = params.visibilities[strided_index];
	    params.visibility_weights[compact_index] = params.visibility_weights[strided_index];
	    params.flags[compact_index] = params.flags[strided_index];
	}
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibilities,params.visibilities,sizeof(std::complex<visibility_base_type>) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.spw_index_array,params.spw_index_array,sizeof(unsigned int) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.visibility_weights,params.visibility_weights,sizeof(visibility_weights_base_type) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flagged_rows,params.flagged_rows,sizeof(bool) * params.row_count,
				     hipMemcpyHostToDevice,compute_stream));
	cudaSafeCall(hipMemcpyAsync(gpu_params.flags,params.flags,sizeof(bool) * params.row_count * params.channel_count,
				     hipMemcpyHostToDevice,compute_stream));
      }
      //invoke computation
      {
	dim3 no_blocks_per_grid(ceil(params.nx / (double)NO_THREADS_PER_BLOCK_DIM),
				ceil(params.ny / (double)NO_THREADS_PER_BLOCK_DIM),
				1);
	dim3 no_threads_per_block(NO_THREADS_PER_BLOCK_DIM,NO_THREADS_PER_BLOCK_DIM,1);
	imaging::grid_single<<<no_blocks_per_grid,no_threads_per_block,0,compute_stream>>>(gpu_params,no_blocks_per_grid,no_threads_per_block);
      }
      //swap buffers device -> host when gridded last chunk
      if (params.is_final_data_chunk){
	gridding_barrier();
	cudaSafeCall(hipMemcpy(params.output_buffer,gpu_params.output_buffer,sizeof(grid_base_type) * params.nx * params.ny,hipMemcpyDeviceToHost));
      }      
      gridding_walltime->stop();
    }
    void facet_single_pol(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_single_pol");
    }
    void grid_duel_pol(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: grid_duel_pol");
    }
    void facet_duel_pol(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_duel_pol");
    }
    void grid_4_cor(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: grid_4_cor");
    }
    void facet_4_cor(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_4_cor");
    }
    void facet_4_cor_corrections(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_4_cor_corrections");
    }
    void grid_sampling_function(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: grid_sampling_function");
    }
    void facet_sampling_function(gridding_parameters & params){
      throw std::runtime_error("Backend Unimplemented exception: facet_sampling_function");
    }
}
